#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
//fireCudaKernel.cu
//
//Kernel and other functions called from the kernel
////////////////////////////////////////////////////////////////////////////////
#include "fireCudaLib.h"

#define Cols (gridDim.x*blockDim.x)
#define Rows (gridDim.y*blockDim.y)

#define DistD (sqrtf(2.)*DistHV)
///////////////////////////////////////////////////////////////////////////////
//
///////////////////////////////////////////////////////////////////////////////

__global__ void FireKernel_SpreadAtNeighbors( float timeNow,
											  											float* ignMap, 
											  											float* timeNext,
											  											float* spread0Map,
											  											float* spreadMaxMap,
											  											float* azimuthMaxMap,
											  											float* eccentricityMap,
											  											float* phiEffWindMap,
											  											float  DistHV,
											  											unsigned int*  lockMap,
																							unsigned int*  lockTime)//11
{

	int nrow, ncol, ncell;
	float ignCell, ignNcell, eccentricity;
	bool looping;
	__shared__ float spreadAny_sh[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float ignTime_sh[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float dir_sh[BLOCK_SIZE][BLOCK_SIZE];

	#define row  ( threadIdx.y + blockIdx.y*blockDim.y)
	#define col  (threadIdx.x + blockIdx.x*blockDim.x)
	#define cell (col + Cols*row)
	
	//Updates ign Map from last iteration
	ignCell = ignMap[cell];

	if( ignCell > timeNow  )
	{
		if (*timeNext > ignCell)
		{	
			looping = true;
			while(looping)
			{
				if (atomicExch(lockTime, 1u) == 0u)
				{
					looping = false;
					if(*timeNext > ignCell)
					{
						*timeNext = ignCell;
					}
					atomicExch(lockTime, 0u);
				}
			}
		}
	}
	else if(ignCell == timeNow )
	{
		//Edge Flag update - N esta a correr com edge flags 
		/*if (row == 0 || row == Rows -1 || col == 0 || col == Cols - 1)
		{
			looping = true;
			while(looping)
			{
				if (atomicExch(lock, 1u) == 0u)
				{
					looping = false;
					*atEdge = 1;
					atomicExch(lock, 0u);
				}
			}
		}*/

		///////////////////////////////////////////////////////////////////////////
		//North Neighbor 
		#define RowN    (-1)
		#define ColN    (0)
		#define azimuth (0)

		nrow = row + RowN;
		ncol = col + ColN;
		ncell = ncol + nrow*Cols;
		ignNcell = ignMap[ncell];

		if(nrow >= 0 && nrow < Rows && ncol >= 0 && ncol < Cols &&
		   ignNcell > timeNow && spreadMaxMap[cell] >= Smidgen)
		{
			//"SpreadAtAzimuth"
			if (phiEffWindMap[cell] < Smidgen && azimuthMaxMap[cell] == azimuth)
				spreadAny_sh[thx][thy] = spreadMaxMap[cell];
			else
			{
				if ((dir_sh[thx][thy] = fabsf(azimuthMaxMap[cell] - azimuth)) > 180)
					dir_sh[thx][thy] = 360. - dir_sh[thx][thy];
			
				dir_sh[thx][thy] = DegToRad(dir_sh[thx][thy]);

				eccentricity = eccentricityMap[cell];
				spreadAny_sh[thx][thy] = __fdividef(spreadMaxMap[cell]*(1-eccentricity),(1- eccentricity*__cosf(dir_sh[thx][thy])));

				if (spreadAny_sh[thx][thy] > INF)
					spreadAny_sh[thx][thy] = spread0Map[cell];
			}//"SpreadAtAzimuth"

			ignTime_sh[thx][thy] = timeNow + __fdividef(DistHV,spreadAny_sh[thx][thy]);

			if(ignTime_sh[thx][thy] < ignNcell)
			{
				looping = true;
				while(looping)
				{
					if (atomicExch(&(lockMap[ncell]), 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < ignMap[ncell])
						{
							ignMap[ncell] =   ignTime_sh[thx][thy];
						}	
						atomicExch(&(lockMap[ncell]), 0u);
					}
				}
			}
			//Update timeNext
			if( ignTime_sh[thx][thy] < *timeNext )
			{	
				looping = true;
				while(looping)
				{
					if (atomicExch(lockTime, 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < *timeNext)
						{
							*timeNext = ignTime_sh[thx][thy];
						}
						atomicExch(lockTime, 0u);
					}
				}
			}	
		}
		
		#undef RowN
		#undef ColN
		#undef azimuth

		///////////////////////////////////////////////////////////////////////////
		//North East Neighbor 
		#define RowN    (-1)
		#define ColN    (1)
		#define azimuth (45.)

		nrow = row + RowN;
		ncol = col + ColN;
		ncell = ncol + nrow*Cols;
		ignNcell = ignMap[ncell];

		if(nrow >= 0 && nrow < Rows && ncol >= 0 && ncol < Cols &&
		   ignNcell > timeNow && spreadMaxMap[cell] >= Smidgen)
		{
			//"SpreadAtAzimuth"
			if (phiEffWindMap[cell] < Smidgen && azimuthMaxMap[cell] == azimuth)
				spreadAny_sh[thx][thy] = spreadMaxMap[cell];
			else
			{
				if ((dir_sh[thx][thy] = fabsf(azimuthMaxMap[cell] - azimuth)) > 180)
					dir_sh[thx][thy] = 360. - dir_sh[thx][thy];
			
				dir_sh[thx][thy] = DegToRad(dir_sh[thx][thy]);

				eccentricity = eccentricityMap[cell];
				spreadAny_sh[thx][thy] = __fdividef(spreadMaxMap[cell]*(1-eccentricity),(1- eccentricity*__cosf(dir_sh[thx][thy])));

				if (spreadAny_sh[thx][thy] > INF)
					spreadAny_sh[thx][thy] = spread0Map[cell];
			}//"SpreadAtAzimuth"

			ignTime_sh[thx][thy] = timeNow + __fdividef(DistHV,spreadAny_sh[thx][thy]);

			if(ignTime_sh[thx][thy] < ignNcell)
			{
				looping = true;
				while(looping)
				{
					if (atomicExch(&(lockMap[ncell]), 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < ignMap[ncell])
						{
							ignMap[ncell] = ignTime_sh[thx][thy];
						}	
						atomicExch(&(lockMap[ncell]), 0u);
					}
				}
			}
			//Update timeNext
			if( ignTime_sh[thx][thy] < *timeNext )
			{	
				looping = true;
				while(looping)
				{
					if (atomicExch(lockTime, 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < *timeNext)
						{
							*timeNext = ignTime_sh[thx][thy];
						}
						atomicExch(lockTime, 0u);
					}
				}
			}	
		}
		
		#undef RowN
		#undef ColN
		#undef azimuth
		
		///////////////////////////////////////////////////////////////////////////
		//East Neighbor 
		#define RowN    (0)
		#define ColN    (1)
		#define azimuth (90.)

		nrow = row + RowN;
		ncol = col + ColN;
		ncell = ncol + nrow*Cols;
		ignNcell = ignMap[ncell];

		if(nrow >= 0 && nrow < Rows && ncol >= 0 && ncol < Cols &&
		   ignNcell > timeNow && spreadMaxMap[cell] >= Smidgen)
		{
			//"SpreadAtAzimuth"
			if (phiEffWindMap[cell] < Smidgen && azimuthMaxMap[cell] == azimuth)
				spreadAny_sh[thx][thy] = spreadMaxMap[cell];
			else
			{
				if ((dir_sh[thx][thy] = fabsf(azimuthMaxMap[cell] - azimuth)) > 180)
					dir_sh[thx][thy] = 360. - dir_sh[thx][thy];
			
				dir_sh[thx][thy] = DegToRad(dir_sh[thx][thy]);

				eccentricity = eccentricityMap[cell];
				spreadAny_sh[thx][thy] = __fdividef(spreadMaxMap[cell]*(1-eccentricity),(1- eccentricity*__cosf(dir_sh[thx][thy])));

				if (spreadAny_sh[thx][thy] > INF)
					spreadAny_sh[thx][thy] = spread0Map[cell];
			}//"SpreadAtAzimuth"

			ignTime_sh[thx][thy] = timeNow + __fdividef(DistHV,spreadAny_sh[thx][thy]);

			if(ignTime_sh[thx][thy] < ignNcell)
			{
				looping = true;
				while(looping)
				{
					if (atomicExch(&(lockMap[ncell]), 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < ignMap[ncell])
						{
							ignMap[ncell] = ignTime_sh[thx][thy];
						}	
						atomicExch(&(lockMap[ncell]), 0u);
					}
				}
			}
			//Update timeNext
			if( ignTime_sh[thx][thy] < *timeNext )
			{	
				looping = true;
				while(looping)
				{
					if (atomicExch(lockTime, 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < *timeNext)
						{
							*timeNext = ignTime_sh[thx][thy];
						}
						atomicExch(lockTime, 0u);
					}
				}
			}	
		}
		
		#undef RowN
		#undef ColN
		#undef azimuth
	
		///////////////////////////////////////////////////////////////////////////
		//South East Neighbor 
		#define RowN    (1)
		#define ColN    (1)
		#define azimuth (135)

		nrow = row + RowN;
		ncol = col + ColN;
		ncell = ncol + nrow*Cols;
		ignNcell = ignMap[ncell];

		if(nrow >= 0 && nrow < Rows && ncol >= 0 && ncol < Cols &&
		   ignNcell > timeNow && spreadMaxMap[cell] >= Smidgen)
		{
			//"SpreadAtAzimuth"
			if (phiEffWindMap[cell] < Smidgen && azimuthMaxMap[cell] == azimuth)
				spreadAny_sh[thx][thy] = spreadMaxMap[cell];
			else
			{
				if ((dir_sh[thx][thy] = fabsf(azimuthMaxMap[cell] - azimuth)) > 180)
					dir_sh[thx][thy] = 360. - dir_sh[thx][thy];
			
				dir_sh[thx][thy] = DegToRad(dir_sh[thx][thy]);

				eccentricity = eccentricityMap[cell];
				spreadAny_sh[thx][thy] = __fdividef(spreadMaxMap[cell]*(1-eccentricity),(1- eccentricity*__cosf(dir_sh[thx][thy])));

				if (spreadAny_sh[thx][thy] > INF)
					spreadAny_sh[thx][thy] = spread0Map[cell];
			}//"SpreadAtAzimuth"

			ignTime_sh[thx][thy] = timeNow + __fdividef(DistHV,spreadAny_sh[thx][thy]);

			if(ignTime_sh[thx][thy] < ignNcell)
			{
				looping = true;
				while(looping)
				{
					if (atomicExch(&(lockMap[ncell]), 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < ignMap[ncell])
						{
							ignMap[ncell] = ignTime_sh[thx][thy];
						}	
						atomicExch(&(lockMap[ncell]), 0u);
					}
				}
			}
			//Update timeNext
			if( ignTime_sh[thx][thy] < *timeNext )
			{	
				looping = true;
				while(looping)
				{
					if (atomicExch(lockTime, 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < *timeNext)
						{
							*timeNext = ignTime_sh[thx][thy];
						}
						atomicExch(lockTime, 0u);
					}
				}
			}	
		}
		
		#undef RowN
		#undef ColN
		#undef azimuth
	
		///////////////////////////////////////////////////////////////////////////
		//South Neighbor 
		#define RowN    (1)
		#define ColN    (0)
		#define azimuth (180.)

		nrow = row + RowN;
		ncol = col + ColN;
		ncell = ncol + nrow*Cols;
		ignNcell = ignMap[ncell];

		if(nrow >= 0 && nrow < Rows && ncol >= 0 && ncol < Cols &&
		   ignNcell > timeNow && spreadMaxMap[cell] >= Smidgen)
		{
			//"SpreadAtAzimuth"
			if (phiEffWindMap[cell] < Smidgen && azimuthMaxMap[cell] == azimuth)
				spreadAny_sh[thx][thy] = spreadMaxMap[cell];
			else
			{
				if ((dir_sh[thx][thy] = fabsf(azimuthMaxMap[cell] - azimuth)) > 180)
					dir_sh[thx][thy] = 360. - dir_sh[thx][thy];
			
				dir_sh[thx][thy] = DegToRad(dir_sh[thx][thy]);

				eccentricity = eccentricityMap[cell];
				spreadAny_sh[thx][thy] = __fdividef(spreadMaxMap[cell]*(1-eccentricity),(1- eccentricity*__cosf(dir_sh[thx][thy])));

				if (spreadAny_sh[thx][thy] > INF)
					spreadAny_sh[thx][thy] = spread0Map[cell];
			}//"SpreadAtAzimuth"

			ignTime_sh[thx][thy] = timeNow + __fdividef(DistHV,spreadAny_sh[thx][thy]);

			if(ignTime_sh[thx][thy] < ignNcell)
			{
				looping = true;
				while(looping)
				{
					if (atomicExch(&(lockMap[ncell]), 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < ignMap[ncell])
						{
							ignMap[ncell] = ignTime_sh[thx][thy];
						}	
						atomicExch(&(lockMap[ncell]), 0u);
					}
				}
			}
			//Update timeNext
			if( ignTime_sh[thx][thy] < *timeNext )
			{	
				looping = true;
				while(looping)
				{
					if (atomicExch(lockTime, 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < *timeNext)
						{
							*timeNext = ignTime_sh[thx][thy];
						}
						atomicExch(lockTime, 0u);
					}
				}
			}	
		}
		#undef RowN
		#undef ColN
		#undef azimuth
	
		///////////////////////////////////////////////////////////////////////////
		//South West Neighbor 
		#define RowN    (1)
		#define ColN    (-1)
		#define azimuth (225)

		
		nrow = row + RowN;
		ncol = col + ColN;
		ncell = ncol + nrow*Cols;
		ignNcell = ignMap[ncell];

		if(nrow >= 0 && nrow < Rows && ncol >= 0 && ncol < Cols &&
		   ignNcell > timeNow && spreadMaxMap[cell] >= Smidgen)
		{
			//"SpreadAtAzimuth"
			if (phiEffWindMap[cell] < Smidgen && azimuthMaxMap[cell] == azimuth)
				spreadAny_sh[thx][thy] = spreadMaxMap[cell];
			else
			{
				if ((dir_sh[thx][thy] = fabsf(azimuthMaxMap[cell] - azimuth)) > 180)
					dir_sh[thx][thy] = 360. - dir_sh[thx][thy];
			
				dir_sh[thx][thy] = DegToRad(dir_sh[thx][thy]);

				eccentricity = eccentricityMap[cell];
				spreadAny_sh[thx][thy] = __fdividef(spreadMaxMap[cell]*(1-eccentricity),(1- eccentricity*__cosf(dir_sh[thx][thy])));

				if (spreadAny_sh[thx][thy] > INF)
					spreadAny_sh[thx][thy] = spread0Map[cell];
			}//"SpreadAtAzimuth"

			ignTime_sh[thx][thy] = timeNow + __fdividef(DistHV,spreadAny_sh[thx][thy]);

			if(ignTime_sh[thx][thy] < ignNcell)
			{
				looping = true;
				while(looping)
				{
					if (atomicExch(&(lockMap[ncell]), 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < ignMap[ncell])
						{
							ignMap[ncell] = ignTime_sh[thx][thy];
						}	
						atomicExch(&(lockMap[ncell]), 0u);
					}
				}
			}
			//Update timeNext
			if( ignTime_sh[thx][thy] < *timeNext )
			{	
				looping = true;
				while(looping)
				{
					if (atomicExch(lockTime, 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < *timeNext)
						{
							*timeNext = ignTime_sh[thx][thy];
						}
						atomicExch(lockTime, 0u);
					}
				}
			}	
		}
		#undef RowN
		#undef ColN
		#undef azimuth
	
		///////////////////////////////////////////////////////////////////////////
		//West Neighbor 
		#define RowN    (0)
		#define ColN    (-1)
		#define azimuth (270.)

		nrow = row + RowN;
		ncol = col + ColN;
		ncell = ncol + nrow*Cols;
		ignNcell = ignMap[ncell];

		if(nrow >= 0 && nrow < Rows && ncol >= 0 && ncol < Cols &&
		   ignNcell > timeNow && spreadMaxMap[cell] >= Smidgen)
		{
			//"SpreadAtAzimuth"
			if (phiEffWindMap[cell] < Smidgen && azimuthMaxMap[cell] == azimuth)
				spreadAny_sh[thx][thy] = spreadMaxMap[cell];
			else
			{
				if ((dir_sh[thx][thy] = fabsf(azimuthMaxMap[cell] - azimuth)) > 180)
					dir_sh[thx][thy] = 360. - dir_sh[thx][thy];
			
				dir_sh[thx][thy] = DegToRad(dir_sh[thx][thy]);

				eccentricity = eccentricityMap[cell];
				spreadAny_sh[thx][thy] = __fdividef(spreadMaxMap[cell]*(1-eccentricity),(1- eccentricity*__cosf(dir_sh[thx][thy])));

				if (spreadAny_sh[thx][thy] > INF)
					spreadAny_sh[thx][thy] = spread0Map[cell];
			}//"SpreadAtAzimuth"

			ignTime_sh[thx][thy] = timeNow + __fdividef(DistHV,spreadAny_sh[thx][thy]);

			if(ignTime_sh[thx][thy] < ignNcell)
			{
				looping = true;
				while(looping)
				{
					if (atomicExch(&(lockMap[ncell]), 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < ignMap[ncell])
						{
							ignMap[ncell] = ignTime_sh[thx][thy];
						}	
						atomicExch(&(lockMap[ncell]), 0u);
					}
				}
			}
			//Update timeNext
			if( ignTime_sh[thx][thy] < *timeNext )
			{	
				looping = true;
				while(looping)
				{
					if (atomicExch(lockTime, 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < *timeNext)
						{
							*timeNext = ignTime_sh[thx][thy];
						}
						atomicExch(lockTime, 0u);
					}
				}
			}	
		}
		#undef RowN
		#undef ColN
		#undef azimuth
	
		///////////////////////////////////////////////////////////////////////////
		//North West Neighbor 
		#define RowN    (-1)
		#define ColN    (-1)
		#define azimuth (315)
		
		nrow = row + RowN;
		ncol = col + ColN;
		ncell = ncol + nrow*Cols;
		ignNcell = ignMap[ncell];

		if(nrow >= 0 && nrow < Rows && ncol >= 0 && ncol < Cols &&
		   ignNcell > timeNow && spreadMaxMap[cell] >= Smidgen)
		{
			//"SpreadAtAzimuth"
			if (phiEffWindMap[cell] < Smidgen && azimuthMaxMap[cell] == azimuth)
				spreadAny_sh[thx][thy] = spreadMaxMap[cell];
			else
			{
				if ((dir_sh[thx][thy] = fabsf(azimuthMaxMap[cell] - azimuth)) > 180)
					dir_sh[thx][thy] = 360. - dir_sh[thx][thy];
			
				dir_sh[thx][thy] = DegToRad(dir_sh[thx][thy]);

				eccentricity = eccentricityMap[cell];
				spreadAny_sh[thx][thy] = __fdividef(spreadMaxMap[cell]*(1-eccentricity),(1- eccentricity*__cosf(dir_sh[thx][thy])));

				if (spreadAny_sh[thx][thy] > INF)
					spreadAny_sh[thx][thy] = spread0Map[cell];
			}//"SpreadAtAzimuth"

			ignTime_sh[thx][thy] = timeNow + __fdividef(DistHV,spreadAny_sh[thx][thy]);

			if(ignTime_sh[thx][thy] < ignNcell)
			{
				looping = true;
				while(looping)
				{
					if (atomicExch(&(lockMap[ncell]), 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < ignMap[ncell])
						{
							ignMap[ncell] = ignTime_sh[thx][thy];
						}	
						atomicExch(&(lockMap[ncell]), 0u);
					}
				}
			}
			//Update timeNext
			if( ignTime_sh[thx][thy] < *timeNext )
			{	
				looping = true;
				while(looping)
				{
					if (atomicExch(lockTime, 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < *timeNext)
						{
							*timeNext = ignTime_sh[thx][thy];
						}
						atomicExch(lockTime, 0u);
					}
				}
			}	
		}
		#undef RowN
		#undef ColN
		#undef azimuth
	
		#if Stencil16
		///////////////////////////////////////////////////////////////////////////
		//a Neighbor 
		#define RowN    (-2)
		#define ColN    (-1)
		#define azimuth (333.43494882292202)
		#define Dist (sqrtf((RowN*RowN + ColN*ColN)*DistHV*DistHV))
		
		nrow = row + RowN;
		ncol = col + ColN;
		ncell = ncol + nrow*Cols;
		ignNcell = ignMap[ncell];

		if(nrow >= 0 && nrow < Rows && ncol >= 0 && ncol < Cols &&
		   ignNcell > timeNow && spreadMaxMap[cell] >= Smidgen)
		{
			//"SpreadAtAzimuth"
			if (phiEffWindMap[cell] < Smidgen && azimuthMaxMap[cell] == azimuth)
				spreadAny_sh[thx][thy] = spreadMaxMap[cell];
			else
			{
				if ((dir_sh[thx][thy] = fabsf(azimuthMaxMap[cell] - azimuth)) > 180)
					dir_sh[thx][thy] = 360. - dir_sh[thx][thy];
			
				dir_sh[thx][thy] = DegToRad(dir_sh[thx][thy]);

				eccentricity = eccentricityMap[cell];
				spreadAny_sh[thx][thy] = __fdividef(spreadMaxMap[cell]*(1-eccentricity),(1- eccentricity*__cosf(dir_sh[thx][thy])));

				if (spreadAny_sh[thx][thy] > INF)
					spreadAny_sh[thx][thy] = spread0Map[cell];
			}//"SpreadAtAzimuth"

			ignTime_sh[thx][thy] = timeNow + __fdividef(DistHV,spreadAny_sh[thx][thy]);

			if(ignTime_sh[thx][thy] < ignNcell)
			{
				looping = true;
				while(looping)
				{
					if (atomicExch(&(lockMap[ncell]), 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < ignMap[ncell])
						{
							ignMap[ncell] = ignTime_sh[thx][thy];
						}	
						atomicExch(&(lockMap[ncell]), 0u);
					}
				}
			}
			//Update timeNext
			if( ignTime_sh[thx][thy] < *timeNext )
			{	
				looping = true;
				while(looping)
				{
					if (atomicExch(lockTime, 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < *timeNext)
						{
							*timeNext = ignTime_sh[thx][thy];
						}
						atomicExch(lockTime, 0u);
					}
				}
			}	
		}
		#undef RowN
		#undef ColN
		#undef azimuth
		#undef Dist
		///////////////////////////////////////////////////////////////////////////
		//b Neighbor 
		#define RowN    (-2)
		#define ColN    (1)
		#define azimuth (26.56505117707799)
		#define Dist (sqrtf((RowN*RowN + ColN*ColN)*DistHV*DistHV))
		
		nrow = row + RowN;
		ncol = col + ColN;
		ncell = ncol + nrow*Cols;
		ignNcell = ignMap[ncell];

		if(nrow >= 0 && nrow < Rows && ncol >= 0 && ncol < Cols &&
		   ignNcell > timeNow && spreadMaxMap[cell] >= Smidgen)
		{
			//"SpreadAtAzimuth"
			if (phiEffWindMap[cell] < Smidgen && azimuthMaxMap[cell] == azimuth)
				spreadAny_sh[thx][thy] = spreadMaxMap[cell];
			else
			{
				if ((dir_sh[thx][thy] = fabsf(azimuthMaxMap[cell] - azimuth)) > 180)
					dir_sh[thx][thy] = 360. - dir_sh[thx][thy];
			
				dir_sh[thx][thy] = DegToRad(dir_sh[thx][thy]);

				eccentricity = eccentricityMap[cell];
				spreadAny_sh[thx][thy] = __fdividef(spreadMaxMap[cell]*(1-eccentricity),(1- eccentricity*__cosf(dir_sh[thx][thy])));

				if (spreadAny_sh[thx][thy] > INF)
					spreadAny_sh[thx][thy] = spread0Map[cell];
			}//"SpreadAtAzimuth"

			ignTime_sh[thx][thy] = timeNow + __fdividef(DistHV,spreadAny_sh[thx][thy]);

			if(ignTime_sh[thx][thy] < ignNcell)
			{
				looping = true;
				while(looping)
				{
					if (atomicExch(&(lockMap[ncell]), 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < ignMap[ncell])
						{
							ignMap[ncell] = ignTime_sh[thx][thy];
						}	
						atomicExch(&(lockMap[ncell]), 0u);
					}
				}
			}
			//Update timeNext
			if( ignTime_sh[thx][thy] < *timeNext )
			{	
				looping = true;
				while(looping)
				{
					if (atomicExch(lockTime, 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < *timeNext)
						{
							*timeNext = ignTime_sh[thx][thy];
						}
						atomicExch(lockTime, 0u);
					}
				}
			}	
		}
		#undef RowN
		#undef ColN
		#undef azimuth
		#undef Dist

		///////////////////////////////////////////////////////////////////////////
		//c Neighbor 
		#define RowN    (-1)
		#define ColN    (-2)
		#define azimuth (296.56505117707798)
		#define Dist (sqrtf((RowN*RowN + ColN*ColN)*DistHV*DistHV))
		
		nrow = row + RowN;
		ncol = col + ColN;
		ncell = ncol + nrow*Cols;
		ignNcell = ignMap[ncell];

		if(nrow >= 0 && nrow < Rows && ncol >= 0 && ncol < Cols &&
		   ignNcell > timeNow && spreadMaxMap[cell] >= Smidgen)
		{
			//"SpreadAtAzimuth"
			if (phiEffWindMap[cell] < Smidgen && azimuthMaxMap[cell] == azimuth)
				spreadAny_sh[thx][thy] = spreadMaxMap[cell];
			else
			{
				if ((dir_sh[thx][thy] = fabsf(azimuthMaxMap[cell] - azimuth)) > 180)
					dir_sh[thx][thy] = 360. - dir_sh[thx][thy];
			
				dir_sh[thx][thy] = DegToRad(dir_sh[thx][thy]);

				eccentricity = eccentricityMap[cell];
				spreadAny_sh[thx][thy] = __fdividef(spreadMaxMap[cell]*(1-eccentricity),(1- eccentricity*__cosf(dir_sh[thx][thy])));

				if (spreadAny_sh[thx][thy] > INF)
					spreadAny_sh[thx][thy] = spread0Map[cell];
			}//"SpreadAtAzimuth"

			ignTime_sh[thx][thy] = timeNow + __fdividef(DistHV,spreadAny_sh[thx][thy]);

			if(ignTime_sh[thx][thy] < ignNcell)
			{
				looping = true;
				while(looping)
				{
					if (atomicExch(&(lockMap[ncell]), 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < ignMap[ncell])
						{
							ignMap[ncell] = ignTime_sh[thx][thy];
						}	
						atomicExch(&(lockMap[ncell]), 0u);
					}
				}
			}
			//Update timeNext
			if( ignTime_sh[thx][thy] < *timeNext )
			{	
				looping = true;
				while(looping)
				{
					if (atomicExch(lockTime, 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < *timeNext)
						{
							*timeNext = ignTime_sh[thx][thy];
						}
						atomicExch(lockTime, 0u);
					}
				}
			}	
		}
		#undef RowN
		#undef ColN
		#undef azimuth
		#undef Dist
		
		///////////////////////////////////////////////////////////////////////////
		//d Neighbor 
		#define RowN    (-1)
		#define ColN    (2)
		#define azimuth (63.43494882292201)
		#define Dist (sqrtf((RowN*RowN + ColN*ColN)*DistHV*DistHV))
		
		nrow = row + RowN;
		ncol = col + ColN;
		ncell = ncol + nrow*Cols;
		ignNcell = ignMap[ncell];

		if(nrow >= 0 && nrow < Rows && ncol >= 0 && ncol < Cols &&
		   ignNcell > timeNow && spreadMaxMap[cell] >= Smidgen)
		{
			//"SpreadAtAzimuth"
			if (phiEffWindMap[cell] < Smidgen && azimuthMaxMap[cell] == azimuth)
				spreadAny_sh[thx][thy] = spreadMaxMap[cell];
			else
			{
				if ((dir_sh[thx][thy] = fabsf(azimuthMaxMap[cell] - azimuth)) > 180)
					dir_sh[thx][thy] = 360. - dir_sh[thx][thy];
			
				dir_sh[thx][thy] = DegToRad(dir_sh[thx][thy]);

				eccentricity = eccentricityMap[cell];
				spreadAny_sh[thx][thy] = __fdividef(spreadMaxMap[cell]*(1-eccentricity),(1- eccentricity*__cosf(dir_sh[thx][thy])));

				if (spreadAny_sh[thx][thy] > INF)
					spreadAny_sh[thx][thy] = spread0Map[cell];
			}//"SpreadAtAzimuth"

			ignTime_sh[thx][thy] = timeNow + __fdividef(DistHV,spreadAny_sh[thx][thy]);

			if(ignTime_sh[thx][thy] < ignNcell)
			{
				looping = true;
				while(looping)
				{
					if (atomicExch(&(lockMap[ncell]), 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < ignMap[ncell])
						{
							ignMap[ncell] = ignTime_sh[thx][thy];
						}	
						atomicExch(&(lockMap[ncell]), 0u);
					}
				}
			}
			//Update timeNext
			if( ignTime_sh[thx][thy] < *timeNext )
			{	
				looping = true;
				while(looping)
				{
					if (atomicExch(lockTime, 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < *timeNext)
						{
							*timeNext = ignTime_sh[thx][thy];
						}
						atomicExch(lockTime, 0u);
					}
				}
			}	
		}
		#undef RowN
		#undef ColN
		#undef azimuth
		#undef Dist
		
		///////////////////////////////////////////////////////////////////////////
		//e Neighbor 
		#define RowN    (1)
		#define ColN    (-2)
		#define azimuth (243.43494882292202)
		#define Dist (sqrtf((RowN*RowN + ColN*ColN)*DistHV*DistHV))
		
		nrow = row + RowN;
		ncol = col + ColN;
		ncell = ncol + nrow*Cols;
		ignNcell = ignMap[ncell];

		if(nrow >= 0 && nrow < Rows && ncol >= 0 && ncol < Cols &&
		   ignNcell > timeNow && spreadMaxMap[cell] >= Smidgen)
		{
			//"SpreadAtAzimuth"
			if (phiEffWindMap[cell] < Smidgen && azimuthMaxMap[cell] == azimuth)
				spreadAny_sh[thx][thy] = spreadMaxMap[cell];
			else
			{
				if ((dir_sh[thx][thy] = fabsf(azimuthMaxMap[cell] - azimuth)) > 180)
					dir_sh[thx][thy] = 360. - dir_sh[thx][thy];
			
				dir_sh[thx][thy] = DegToRad(dir_sh[thx][thy]);

				eccentricity = eccentricityMap[cell];
				spreadAny_sh[thx][thy] = __fdividef(spreadMaxMap[cell]*(1-eccentricity),(1- eccentricity*__cosf(dir_sh[thx][thy])));

				if (spreadAny_sh[thx][thy] > INF)
					spreadAny_sh[thx][thy] = spread0Map[cell];
			}//"SpreadAtAzimuth"

			ignTime_sh[thx][thy] = timeNow + __fdividef(DistHV,spreadAny_sh[thx][thy]);

			if(ignTime_sh[thx][thy] < ignNcell)
			{
				looping = true;
				while(looping)
				{
					if (atomicExch(&(lockMap[ncell]), 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < ignMap[ncell])
						{
							ignMap[ncell] = ignTime_sh[thx][thy];
						}	
						atomicExch(&(lockMap[ncell]), 0u);
					}
				}
			}
			//Update timeNext
			if( ignTime_sh[thx][thy] < *timeNext )
			{	
				looping = true;
				while(looping)
				{
					if (atomicExch(lockTime, 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < *timeNext)
						{
							*timeNext = ignTime_sh[thx][thy];
						}
						atomicExch(lockTime, 0u);
					}
				}
			}	
		}
		#undef RowN
		#undef ColN
		#undef azimuth
		#undef Dist
		
		///////////////////////////////////////////////////////////////////////////
		//f Neighbor 
		#define RowN    (1)
		#define ColN    (2)
		#define azimuth (116.56505117707799)
		#define Dist (sqrtf((RowN*RowN + ColN*ColN)*DistHV*DistHV))
		
		nrow = row + RowN;
		ncol = col + ColN;
		ncell = ncol + nrow*Cols;
		ignNcell = ignMap[ncell];

		if(nrow >= 0 && nrow < Rows && ncol >= 0 && ncol < Cols &&
		   ignNcell > timeNow && spreadMaxMap[cell] >= Smidgen)
		{
			//"SpreadAtAzimuth"
			if (phiEffWindMap[cell] < Smidgen && azimuthMaxMap[cell] == azimuth)
				spreadAny_sh[thx][thy] = spreadMaxMap[cell];
			else
			{
				if ((dir_sh[thx][thy] = fabsf(azimuthMaxMap[cell] - azimuth)) > 180)
					dir_sh[thx][thy] = 360. - dir_sh[thx][thy];
			
				dir_sh[thx][thy] = DegToRad(dir_sh[thx][thy]);

				eccentricity = eccentricityMap[cell];
				spreadAny_sh[thx][thy] = __fdividef(spreadMaxMap[cell]*(1-eccentricity),(1- eccentricity*__cosf(dir_sh[thx][thy])));

				if (spreadAny_sh[thx][thy] > INF)
					spreadAny_sh[thx][thy] = spread0Map[cell];
			}//"SpreadAtAzimuth"

			ignTime_sh[thx][thy] = timeNow + __fdividef(DistHV,spreadAny_sh[thx][thy]);

			if(ignTime_sh[thx][thy] < ignNcell)
			{
				looping = true;
				while(looping)
				{
					if (atomicExch(&(lockMap[ncell]), 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < ignMap[ncell])
						{
							ignMap[ncell] = ignTime_sh[thx][thy];
						}	
						atomicExch(&(lockMap[ncell]), 0u);
					}
				}
			}
			//Update timeNext
			if( ignTime_sh[thx][thy] < *timeNext )
			{	
				looping = true;
				while(looping)
				{
					if (atomicExch(lockTime, 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < *timeNext)
						{
							*timeNext = ignTime_sh[thx][thy];
						}
						atomicExch(lockTime, 0u);
					}
				}
			}	
		}
		#undef RowN
		#undef ColN
		#undef azimuth
		#undef Dist
		
		///////////////////////////////////////////////////////////////////////////
		//g Neighbor 
		#define RowN    (2)
		#define ColN    (-1)
		#define azimuth (206.56505117707798)
		#define Dist (sqrtf((RowN*RowN + ColN*ColN)*DistHV*DistHV))
		
		nrow = row + RowN;
		ncol = col + ColN;
		ncell = ncol + nrow*Cols;
		ignNcell = ignMap[ncell];

		if(nrow >= 0 && nrow < Rows && ncol >= 0 && ncol < Cols &&
		   ignNcell > timeNow && spreadMaxMap[cell] >= Smidgen)
		{
			//"SpreadAtAzimuth"
			if (phiEffWindMap[cell] < Smidgen && azimuthMaxMap[cell] == azimuth)
				spreadAny_sh[thx][thy] = spreadMaxMap[cell];
			else
			{
				if ((dir_sh[thx][thy] = fabsf(azimuthMaxMap[cell] - azimuth)) > 180)
					dir_sh[thx][thy] = 360. - dir_sh[thx][thy];
			
				dir_sh[thx][thy] = DegToRad(dir_sh[thx][thy]);

				eccentricity = eccentricityMap[cell];
				spreadAny_sh[thx][thy] = __fdividef(spreadMaxMap[cell]*(1-eccentricity),(1- eccentricity*__cosf(dir_sh[thx][thy])));

				if (spreadAny_sh[thx][thy] > INF)
					spreadAny_sh[thx][thy] = spread0Map[cell];
			}//"SpreadAtAzimuth"

			ignTime_sh[thx][thy] = timeNow + __fdividef(DistHV,spreadAny_sh[thx][thy]);

			if(ignTime_sh[thx][thy] < ignNcell)
			{
				looping = true;
				while(looping)
				{
					if (atomicExch(&(lockMap[ncell]), 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < ignMap[ncell])
						{
							ignMap[ncell] = ignTime_sh[thx][thy];
						}	
						atomicExch(&(lockMap[ncell]), 0u);
					}
				}
			}
			//Update timeNext
			if( ignTime_sh[thx][thy] < *timeNext )
			{	
				looping = true;
				while(looping)
				{
					if (atomicExch(lockTime, 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < *timeNext)
						{
							*timeNext = ignTime_sh[thx][thy];
						}
						atomicExch(lockTime, 0u);
					}
				}
			}	
		}
		#undef RowN
		#undef ColN
		#undef azimuth
		#undef Dist
		
		///////////////////////////////////////////////////////////////////////////
		//h Neighbor 
		#define RowN    (2)
		#define ColN    (1)
		#define azimuth (153.43494882292202)
		#define Dist (sqrtf((RowN*RowN + ColN*ColN)*DistHV*DistHV))
		
		nrow = row + RowN;
		ncol = col + ColN;
		ncell = ncol + nrow*Cols;
		ignNcell = ignMap[ncell];

		if(nrow >= 0 && nrow < Rows && ncol >= 0 && ncol < Cols &&
		   ignNcell > timeNow && spreadMaxMap[cell] >= Smidgen)
		{
			//"SpreadAtAzimuth"
			if (phiEffWindMap[cell] < Smidgen && azimuthMaxMap[cell] == azimuth)
				spreadAny_sh[thx][thy] = spreadMaxMap[cell];
			else
			{
				if ((dir_sh[thx][thy] = fabsf(azimuthMaxMap[cell] - azimuth)) > 180)
					dir_sh[thx][thy] = 360. - dir_sh[thx][thy];
			
				dir_sh[thx][thy] = DegToRad(dir_sh[thx][thy]);

				eccentricity = eccentricityMap[cell];
				spreadAny_sh[thx][thy] = __fdividef(spreadMaxMap[cell]*(1-eccentricity),(1- eccentricity*__cosf(dir_sh[thx][thy])));

				if (spreadAny_sh[thx][thy] > INF)
					spreadAny_sh[thx][thy] = spread0Map[cell];
			}//"SpreadAtAzimuth"

			ignTime_sh[thx][thy] = timeNow + __fdividef(DistHV,spreadAny_sh[thx][thy]);

			if(ignTime_sh[thx][thy] < ignNcell)
			{
				looping = true;
				while(looping)
				{
					if (atomicExch(&(lockMap[ncell]), 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < ignMap[ncell])
						{
							ignMap[ncell] = ignTime_sh[thx][thy];
						}	
						atomicExch(&(lockMap[ncell]), 0u);
					}
				}
			}
			//Update timeNext
			if( ignTime_sh[thx][thy] < *timeNext )
			{	
				looping = true;
				while(looping)
				{
					if (atomicExch(lockTime, 1u) == 0u)
					{
						looping = false;
						if(ignTime_sh[thx][thy] < *timeNext)
						{
							*timeNext = ignTime_sh[thx][thy];
						}
						atomicExch(lockTime, 0u);
					}
				}
			}	
		}
		#undef RowN
		#undef ColN
		#undef azimuth
		#undef Dist

		#endif //For Stencil 16
	}	

}
